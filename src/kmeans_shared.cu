#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>
#include <iomanip>
#include <cfloat>
#include <hip/hip_runtime.h>

#include <chrono>

using namespace std;

struct Point {
    int id;               // Point ID
    int assignedCluster;   // Cluster ID
    int numDimensions;     // Number of dimensions
    float* coordinates;    // Raw pointer for values of each dimension
};

struct Cluster {
    int id;                // Cluster ID
    float* centroid;       // Raw pointer for centroid of the cluster
};

static unsigned long int k_next = 1;
static unsigned long kmeans_rmax = 32767;

int kmeans_rand() {
    k_next = k_next * 1103515245 + 12345;
    return (unsigned int)(k_next / 65536) % (kmeans_rmax + 1);
}

void kmeans_srand(unsigned int seed) {
    k_next = seed;
}

void initializePoint(Point& point, int id, const string& line, int numDimensions) {
    point.id = id;
    point.assignedCluster = -1;  // Not assigned to any cluster yet
    point.coordinates = new float[numDimensions];  // Dynamically allocate array

    stringstream ss(line);
    int skipInt;
    float value;

    ss >> skipInt;  // Skip the first integer value

    for (int d = 0; d < numDimensions; d++) {
        if (ss >> value) {
            point.coordinates[d] = value;
        }
    }

    point.numDimensions = numDimensions;
}

void initializeClusters(vector<Cluster>& clusters, const vector<Point>& points, int numClusters, int numDimensions) {
    for (int i = 0; i < numClusters; i++) {
        Cluster cluster;
        cluster.centroid = new float[numDimensions];  // Dynamically allocate array for centroid

        int idx = kmeans_rand() % points.size();  

        for (int d = 0; d < numDimensions; d++) {
            cluster.centroid[d] = points[idx].coordinates[d];  // Assign the centroid from a point
        }
        clusters.push_back(cluster);
    }
}

// Cuda kernel using shared mem to assign each point to nearest cluster.
__global__ void assignPointsToClusters(Point* points, Cluster* clusters, int numPoints, int numClusters, int numDimensions) {
    extern __shared__ float sharedCentroids[];  // Dynamically allocated shared memory for centroids

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load cluster centroids into shared memory for all threads
    for (int c = threadIdx.x; c < numClusters * numDimensions; c += blockDim.x) {
        sharedCentroids[c] = clusters[0].centroid[c];
    }

    // Synchronize to ensure all centroids are loaded
    __syncthreads();

    if (idx < numPoints) {
        float minDistance = FLT_MAX;
        int nearestCluster = -1;

        // Calculate the distance to each centroid
        for (int c = 0; c < numClusters; c++) {
            float dist = 0.0f;
            for (int d = 0; d < numDimensions; d++) {
                float diff = points[idx].coordinates[d] - sharedCentroids[c * numDimensions + d];
                dist += diff * diff;
            }

            if (dist < minDistance) {
                minDistance = dist;
                nearestCluster = c;
            }
        }
        points[idx].assignedCluster = nearestCluster;
    }
}

// Cuda Kernel using shared memory to yodate the centroids of each cluster
__global__ void updateCentroids(Point* points, Cluster* clusters, int numPoints, int numClusters, int numDimensions) {
    extern __shared__ float sharedCentroidSum[];  // Dynamically allocated shared memory for sums

    int clusterIdx = blockIdx.x;  // One block per cluster
    int pointIdx = threadIdx.x;

    // Initialize shared memory for centroids to zero
    for (int d = threadIdx.x; d < numDimensions; d += blockDim.x) {
        sharedCentroidSum[d] = 0.0f;
    }
    __syncthreads();

    // Perform parallel reduction to sum points assigned to this cluster
    for (int i = pointIdx; i < numPoints; i += blockDim.x) {
        if (points[i].assignedCluster == clusterIdx) {
            for (int d = 0; d < numDimensions; d++) {
                sharedCentroidSum[d] += points[i].coordinates[d];
            }
        }
    }
    __syncthreads();

    // Reduction step: sum within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            for (int d = 0; d < numDimensions; d++) {
                sharedCentroidSum[d] += sharedCentroidSum[threadIdx.x + stride];
            }
        }
        __syncthreads();
    }

    // Normalize and update the centroids in global memory
    if (threadIdx.x == 0) {
        int numAssignedPoints = 0;
        for (int i = 0; i < numPoints; i++) {
            if (points[i].assignedCluster == clusterIdx) {
                numAssignedPoints++;
            }
        }

        if (numAssignedPoints > 0) {
            for (int d = 0; d < numDimensions; d++) {
                clusters[clusterIdx].centroid[d] = sharedCentroidSum[d] / numAssignedPoints;
            }
        }
    }
}


// kmeans function for shared mem implementation
void kmeans_cuda(Point* h_points, Cluster* h_clusters, int numPoints, int numClusters, int numDimensions, int maxIterations, float convergenceThreshold) {
    Point* d_points;
    Cluster* d_clusters;

    // Allocate device memory for Points and Clusters
    hipMalloc(&d_points, numPoints * sizeof(Point));
    hipMalloc(&d_clusters, numClusters * sizeof(Cluster));

    // Create CUDA events for timing
    hipEvent_t startTransferToGPU, endTransferToGPU, startKernel, endKernel, startTransferFromGPU, endTransferFromGPU;
    hipEventCreate(&startTransferToGPU);
    hipEventCreate(&endTransferToGPU);
    hipEventCreate(&startKernel);
    hipEventCreate(&endKernel);
    hipEventCreate(&startTransferFromGPU);
    hipEventCreate(&endTransferFromGPU);

    // Record the start time for transfer to GPU
    hipEventRecord(startTransferToGPU);

    // Copy points and clusters from host (CPU) to device (GPU)
    hipMemcpy(d_points, h_points, numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_clusters, h_clusters, numClusters * sizeof(Cluster), hipMemcpyHostToDevice);

    // Record the end time for transfer to GPU
    hipEventRecord(endTransferToGPU);
    hipEventSynchronize(endTransferToGPU);

    float timeTransferToGPU;
    hipEventElapsedTime(&timeTransferToGPU, startTransferToGPU, endTransferToGPU);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;

    // Record the start time for the kernel execution
    hipEventRecord(startKernel);

    // Main loop: run maxIterations times
    for (int iter = 0; iter < maxIterations; iter++) {
        assignPointsToClusters<<<blocksPerGrid, threadsPerBlock>>>(d_points, d_clusters, numPoints, numClusters, numDimensions);
        updateCentroids<<<numClusters, threadsPerBlock>>>(d_points, d_clusters, numPoints, numClusters, numDimensions);
        hipDeviceSynchronize();
    }

    // Record the end time for the kernel execution
    hipEventRecord(endKernel);
    hipEventSynchronize(endKernel);

    float timeKernel;
    hipEventElapsedTime(&timeKernel, startKernel, endKernel);

    // Record the start time for transfer from GPU
    hipEventRecord(startTransferFromGPU);

    // Copy points and clusters back from device (GPU) to host (CPU)
    hipMemcpy(h_points, d_points, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(h_clusters, d_clusters, numClusters * sizeof(Cluster), hipMemcpyDeviceToHost);

    // Record the end time for transfer from GPU
    hipEventRecord(endTransferFromGPU);
    hipEventSynchronize(endTransferFromGPU);

    float timeTransferFromGPU;
    hipEventElapsedTime(&timeTransferFromGPU, startTransferFromGPU, endTransferFromGPU);

    // Free device memory
    hipFree(d_points);
    hipFree(d_clusters);

    // Print out the times
    cout << "Time spent transferring data to GPU: " << timeTransferToGPU << " ms" << endl;
    cout << "Time spent executing kernels: " << timeKernel << " ms" << endl;
    cout << "Time spent transferring data from GPU: " << timeTransferFromGPU << " ms" << endl;

    // Cleanup CUDA events
    hipEventDestroy(startTransferToGPU);
    hipEventDestroy(endTransferToGPU);
    hipEventDestroy(startKernel);
    hipEventDestroy(endKernel);
    hipEventDestroy(startTransferFromGPU);
    hipEventDestroy(endTransferFromGPU);
}



// Main function
int main(int argc, char* argv[]) {
    int numClusters = 0, numDimensions = 0, maxIterations = 150, seed = 0;
    string inputFilename;
    float convergenceThreshold = 1e-5;
    bool printCentroids = false;

    // Parse command-line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-k") == 0) numClusters = atoi(argv[++i]);
        else if (strcmp(argv[i], "-d") == 0) numDimensions = atoi(argv[++i]);
        else if (strcmp(argv[i], "-i") == 0) inputFilename = argv[++i];
        else if (strcmp(argv[i], "-m") == 0) maxIterations = atoi(argv[++i]);
        else if (strcmp(argv[i], "-t") == 0) convergenceThreshold = atof(argv[++i]);
        else if (strcmp(argv[i], "-s") == 0) seed = atoi(argv[++i]);
        else if (strcmp(argv[i], "-c") == 0) printCentroids = true;
    }

    vector<Point> points;
    ifstream infile(inputFilename);
    string line;

    // Read the total number of points
    int totalPoints;
    if (getline(infile, line)) {
        totalPoints = stoi(line);
    } else {
        cerr << "Error: Unable to read the number of points." << endl;
        return 1;
    }

    // Read each point
    int pointId = 0;
    while (getline(infile, line) && pointId < totalPoints) {
        Point point;
        initializePoint(point, pointId, line, numDimensions);
        points.push_back(point);
        pointId++;
    }
    infile.close();

    vector<Cluster> clusters;
    kmeans_srand(seed);
    initializeClusters(clusters, points, numClusters, numDimensions);
    
    // Run the CUDA implementation
    auto start = chrono::high_resolution_clock::now();
    kmeans_cuda(points.data(), clusters.data(), totalPoints, numClusters, numDimensions, maxIterations, convergenceThreshold);
    auto end = chrono::high_resolution_clock::now();

    double totalTime = chrono::duration_cast<chrono::milliseconds>(end - start).count();
    double timePerIteration = totalTime / maxIterations;


    cout << maxIterations << "," << fixed << setprecision(6) << timePerIteration << endl;
    if (printCentroids) {
        for (int clusterId = 0; clusterId < numClusters; clusterId++) {
            cout << clusterId << " ";
            for (int d = 0; d < numDimensions; d++) {
                cout << fixed << setprecision(5) << clusters[clusterId].centroid[d] << " ";
            }
            cout << endl;
        }
    } else {
        cout << "clusters:";
        for (const auto& point : points) {
            cout << " " << point.assignedCluster;
        }
        cout << endl;
    }

    for (auto& point : points) {
        delete[] point.coordinates;
    }
    for (auto& cluster : clusters) {
        delete[] cluster.centroid;
    }

    return 0;
}
